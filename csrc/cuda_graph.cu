#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAGraph.h>

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#define CU_CALL(x) do { hipError_t result = x; if (result != hipSuccess) { printf("CUDA error: %d\n", result); return; } } while(0)

struct CUDAGraph_t : public at::cuda::CUDAGraph {
  hipGraph_t get_graph() {
    return graph_;
  }

  void set_graph(hipGraph_t &graph) {
    graph_ = graph;
  }

  void set_graph_exec(hipGraphExec_t &graph_exec) {
    graph_exec_ = graph_exec;
  }

  int get_capture_dev() {
    return capture_dev_;
  }
};

std::vector<uint64_t> kernelParams;

void loadKernelNodeParams(const hipKernelNodeParams &in, hipKernelNodeParams &out) {
  printf("kernel node params:\n");
  printf("ctx: %p\n", in.ctx);
  printf("extra: %p\n", in.extra);
  printf("func: %p\n", in.func);
  printf("kern: %p\n", in.kern);
  printf("kernelParams: %p\n", in.kernelParams);

  kernelParams.push_back((uint64_t)in.kernelParams);

  out = in;
  // we don't set extra field
  if (in.extra) {
    printf("extra is not nil\n");
  }
  // we don't set kern, if func is not null
  if (in.func == nullptr) {
    printf("func is nil\n");
  } else {
    out.kern = nullptr;
  }
}

void loadMemcpyNodeParams(const HIP_MEMCPY3D &in, HIP_MEMCPY3D &out) {
  printf("memcpy node params:\n");
  printf("srcXInBytes: %d\n", in.srcXInBytes);
  printf("srcY: %d\n", in.srcY);
  printf("srcZ: %d\n", in.srcZ);
  printf("srcLOD: %d\n", in.srcLOD);

  out = in;
}

struct CUgraphNodeParam {
  hipGraphNodeType type;

  union {
    hipKernelNodeParams kernel_params;
    HIP_MEMCPY3D memcpy_params;
  } p;
};

#define MAX_NODE_AND_EDGE_NUM 600

void save_cuda_graph (
  at::cuda::CUDAGraph& g
  ) {
  CUDAGraph_t *g_ = (CUDAGraph_t*)(&g);
  const hipGraph_t graph_ = g_->get_graph();
  CU_CALL(hipGraphDebugDotPrint(graph_, "cuda_graph.dot", 1));

  size_t numCUGraphNodes = MAX_NODE_AND_EDGE_NUM;
  hipGraphNode_t nodes[MAX_NODE_AND_EDGE_NUM];
  CU_CALL(hipGraphGetNodes(graph_, nodes, &numCUGraphNodes));
  printf("cuda graph nodes: %d\n", numCUGraphNodes);

  std::vector<CUgraphNodeParam> saved_nodes_params;

  std::map<hipGraphNode_t, int> node_to_idx;

  for (int i = 0; i < numCUGraphNodes; i++) {
    hipGraphNode_t node = nodes[i];
    hipGraphNodeType type;
    CU_CALL(hipGraphNodeGetType(node, &type));
    printf("node idx: %d, node type: %d\n", i, type);

    CUgraphNodeParam param;
    param.type = type;

    node_to_idx.insert(std::make_pair(node, i));

    switch (type) {
      case hipGraphNodeTypeKernel: {
        hipKernelNodeParams pNodeParams;
        CU_CALL(hipGraphKernelNodeGetParams(node, &pNodeParams));
        param.p.kernel_params = pNodeParams;
        break;
      }
      case hipGraphNodeTypeMemcpy: {
        HIP_MEMCPY3D pNodeParams;
        CU_CALL(hipDrvGraphMemcpyNodeGetParams(node, &pNodeParams));
        param.p.memcpy_params = pNodeParams;
        break;
      }
      default: {
        printf("error node type: %d\n", type);
        break;
      }
    }

    saved_nodes_params.push_back(param);
  }

  hipGraphNode_t from[MAX_NODE_AND_EDGE_NUM];
  hipGraphNode_t to[MAX_NODE_AND_EDGE_NUM];
  size_t savedEdges = MAX_NODE_AND_EDGE_NUM;
  CU_CALL(hipGraphGetEdges(graph_, from, to, &savedEdges));

  int saved_from_idx[MAX_NODE_AND_EDGE_NUM];
  int saved_to_idx[MAX_NODE_AND_EDGE_NUM];
  for (int i = 0; i < savedEdges; i++) {
    int from_idx = node_to_idx[from[i]];
    int to_idx = node_to_idx[to[i]];
    saved_from_idx[i] = from_idx;
    saved_to_idx[i] = to_idx;
  }

  printf("=============== saved edges: %d\n", savedEdges);

  // ===========================================================

  // Create an new CUDA graph
  // hipGraph_t newGraph;
  // CU_CALL(hipGraphCreate(&newGraph, 0));

  // std::vector<hipGraphNode_t> new_nodes_vec;

  // hipCtx_t ctx;
  // CU_CALL(hipDevicePrimaryCtxRetain(&ctx, 0));

  // for (int i = 0; i < saved_nodes_params.size(); i++) {
  //   printf("===============================\n");
    
  //   CUgraphNodeParam param = saved_nodes_params[i];
  //   hipGraphNode_t newNode;

  //   switch (param.type) {
  //     case hipGraphNodeTypeKernel: {
  //       hipKernelNodeParams p;
  //       loadKernelNodeParams(param.p.kernel_params, p);
  //       CU_CALL(hipGraphAddKernelNode(&newNode, newGraph, nullptr, 0, &p));
  //       new_nodes_vec.push_back(newNode);
  //       break;
  //     }
  //     case hipGraphNodeTypeMemcpy: {
  //       HIP_MEMCPY3D p;
  //       loadMemcpyNodeParams(param.p.memcpy_params, p);
  //       CU_CALL(hipDrvGraphAddMemcpyNode(&newNode, newGraph, nullptr, 0, &p, ctx));
  //       new_nodes_vec.push_back(newNode);
  //       break;
  //     }
  //     default: {
  //       printf("error node type: %d\n", param.type);
  //       break;
  //     }
  //   }
  // }

  // // uint64_t minKernelParams = 0;
  // // for (int i = 0; i < kernelParams.size(); i++) {
  // //   if (minKernelParams == 0 || kernelParams[i] < minKernelParams) {
  // //     minKernelParams = kernelParams[i];
  // //   }
  // // }
  // // for (int i = 0; i < kernelParams.size(); i++) {
  // //   printf("kernel idx: %d, kernel params offset: %ld\n", i, kernelParams[i] - minKernelParams);
  // // }

  // hipGraphNode_t saved_from[MAX_NODE_AND_EDGE_NUM];
  // hipGraphNode_t saved_to[MAX_NODE_AND_EDGE_NUM];

  // for (int i = 0; i < savedEdges; i++) {
  //   saved_from[i] = new_nodes_vec[saved_from_idx[i]];
  //   saved_to[i] = new_nodes_vec[saved_to_idx[i]];
  // }

  // CU_CALL(hipGraphAddDependencies(newGraph, saved_from, saved_to, savedEdges));

  // g_->set_graph(newGraph);

  // hipGraphExec_t newGraphExec;
  // CU_CALL(hipGraphInstantiate(&newGraphExec, newGraph, 0));
  // g_->set_graph_exec(newGraphExec);
}
